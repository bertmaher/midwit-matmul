#include "hip/hip_runtime.h"
#include "wgmma_kernel.h"

#include <hip/hip_bf16.h>

#include <cassert>
#include <cstdint>


__device__ uint64_t desc(uint64_t addr, uint64_t leading_offset, uint64_t stride_offset, uint64_t base_offset, uint64_t swizzle) {
  return
    ((addr & 0x3FFFF) >> 4) |
    ((leading_offset >> 4) << 16) |
    ((stride_offset >> 4) << 32) |
    (base_offset << 49) |
    (swizzle << 62);
}

__device__ void wgmma(float r[128], uint64_t desc_a, uint64_t desc_b) {
  int scale_d = false;
  int imm_scale_a = 1;
  int imm_scale_b = 1;
  int imm_trans_a = 0;
  int imm_trans_b = 0;
  asm volatile(
      ".reg .pred %%p;\n\t" \
      "setp.eq.s32 %%p, %130, 1;\n\t" \
      "wgmma.mma_async.sync.aligned.m64n256k16.f32.bf16.bf16 " \
      "{%0, %1, %2, %3, %4, %5, %6, %7, %8, %9, %10, %11, %12, %13, %14, %15, %16, %17, %18, %19, %20, %21, %22, %23, %24, %25, %26, %27, %28, %29, %30, %31, %32, %33, %34, %35, %36, %37, %38, %39, %40, %41, %42, %43, %44, %45, %46, %47, %48, %49, %50, %51, %52, %53, %54, %55, %56, %57, %58, %59, %60, %61, %62, %63, %64, %65, %66, %67, %68, %69, %70, %71, %72, %73, %74, %75, %76, %77, %78, %79, %80, %81, %82, %83, %84, %85, %86, %87, %88, %89, %90, %91, %92, %93, %94, %95, %96, %97, %98, %99, %100, %101, %102, %103, %104, %105, %106, %107, %108, %109, %110, %111, %112, %113, %114, %115, %116, %117, %118, %119, %120, %121, %122, %123, %124, %125, %126, %127}, " \
      "%128, " \
      "%129, " \
      "%%p, 1, 1, 0, 0;"
      : "=f"(r[0]), "=f"(r[1]), "=f"(r[2]), "=f"(r[3]), "=f"(r[4]), "=f"(r[5]), "=f"(r[6]), "=f"(r[7]), "=f"(r[8]), "=f"(r[9]), "=f"(r[10]), "=f"(r[11]), "=f"(r[12]), "=f"(r[13]), "=f"(r[14]), "=f"(r[15]), "=f"(r[16]), "=f"(r[17]), "=f"(r[18]), "=f"(r[19]), "=f"(r[20]), "=f"(r[21]), "=f"(r[22]), "=f"(r[23]), "=f"(r[24]), "=f"(r[25]), "=f"(r[26]), "=f"(r[27]), "=f"(r[28]), "=f"(r[29]), "=f"(r[30]), "=f"(r[31]), "=f"(r[32]), "=f"(r[33]), "=f"(r[34]), "=f"(r[35]), "=f"(r[36]), "=f"(r[37]), "=f"(r[38]), "=f"(r[39]), "=f"(r[40]), "=f"(r[41]), "=f"(r[42]), "=f"(r[43]), "=f"(r[44]), "=f"(r[45]), "=f"(r[46]), "=f"(r[47]), "=f"(r[48]), "=f"(r[49]), "=f"(r[50]), "=f"(r[51]), "=f"(r[52]), "=f"(r[53]), "=f"(r[54]), "=f"(r[55]), "=f"(r[56]), "=f"(r[57]), "=f"(r[58]), "=f"(r[59]), "=f"(r[60]), "=f"(r[61]), "=f"(r[62]), "=f"(r[63]), "=f"(r[64]), "=f"(r[65]), "=f"(r[66]), "=f"(r[67]), "=f"(r[68]), "=f"(r[69]), "=f"(r[70]), "=f"(r[71]), "=f"(r[72]), "=f"(r[73]), "=f"(r[74]), "=f"(r[75]), "=f"(r[76]), "=f"(r[77]), "=f"(r[78]), "=f"(r[79]), "=f"(r[80]), "=f"(r[81]), "=f"(r[82]), "=f"(r[83]), "=f"(r[84]), "=f"(r[85]), "=f"(r[86]), "=f"(r[87]), "=f"(r[88]), "=f"(r[89]), "=f"(r[90]), "=f"(r[91]), "=f"(r[92]), "=f"(r[93]), "=f"(r[94]), "=f"(r[95]), "=f"(r[96]), "=f"(r[97]), "=f"(r[98]), "=f"(r[99]), "=f"(r[100]), "=f"(r[101]), "=f"(r[102]), "=f"(r[103]), "=f"(r[104]), "=f"(r[105]), "=f"(r[106]), "=f"(r[107]), "=f"(r[108]), "=f"(r[109]), "=f"(r[110]), "=f"(r[111]), "=f"(r[112]), "=f"(r[113]), "=f"(r[114]), "=f"(r[115]), "=f"(r[116]), "=f"(r[117]), "=f"(r[118]), "=f"(r[119]), "=f"(r[120]), "=f"(r[121]), "=f"(r[122]), "=f"(r[123]), "=f"(r[124]), "=f"(r[125]), "=f"(r[126]), "=f"(r[127])
      : "l"(desc_a), "l"(desc_b), "r"(scale_d), "r"(imm_scale_a), "r"(imm_scale_b), "r"(imm_trans_a), "r"(imm_trans_b)
  );
}

__device__ void wgmma_commit_group() {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  asm volatile("wgmma.commit_group.sync.aligned;");
#endif
}

__device__ void wgmma_wait_group() {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  asm volatile("wgmma.wait_group.sync.aligned 1;");
#endif
}

__global__ void mma_wgmma(__hip_bfloat16* a, __hip_bfloat16* b, __hip_bfloat16* c) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
  constexpr int a_size = 64 * 16;
  constexpr int b_size = 256 * 16;
  __shared__ __hip_bfloat16 a_shared[64 * 16];
  __shared__ __hip_bfloat16 b_shared[256 * 16];

  auto tid = threadIdx.x;
  auto bdim = blockDim.x;

  for (int i = tid; i < a_size; i += bdim) {
    a_shared[i] = a[i];
  }
  for (int i = tid; i < b_size; i += bdim) {
    b_shared[i] = b[i];
  }

  float c_regs[128] = {0.0f};
  wgmma(c_regs, desc((uint64_t)a_shared, 128, 256, 0, 0), desc((uint64_t)b_shared, 128 * 256 / 8, 128, 0, 0));
  wgmma_commit_group();
  wgmma_wait_group();

  for (int i = 0; i < 128; i++) {
    c[tid * bdim + i] = __float2bfloat16(c_regs[i]);
  }
#endif
}


__global__ void mma_naive(__hip_bfloat16* a, __hip_bfloat16* b, __hip_bfloat16* c) {
  constexpr int a_size = 64 * 16;
  constexpr int b_size = 256 * 16;
  __shared__ __hip_bfloat16 a_shared[64 * 16];
  __shared__ __hip_bfloat16 b_shared[256 * 16];

  auto tid = threadIdx.x;
  auto bdim = blockDim.x;

  for (int i = tid; i < a_size; i += bdim) {
    a_shared[i] = a[i];
  }
  for (int i = tid; i < b_size; i += bdim) {
    b_shared[i] = b[i];
  }

  if (tid == 0) {
    for (int m = 0; m < 64; m++) {
      for (int n = 0; n < 256; n++) {
        float p = 0.0;
        for (int k = 0; k < 16; k++) {
          p += __bfloat162float(a_shared[m * 16 + k]) * __bfloat162float(b_shared[n * 16 + k]);
        }
        c[m * 256 + n] = __float2bfloat16(p);
      }
    }
  }
}

void wgmma_kernel(void* a, void* b, void* c, int m, int n, int k) {
  assert(m == 64);
  assert(n == 256);
  assert(k == 16);

  mma_wgmma<<<1, 128>>>((__hip_bfloat16*)a, (__hip_bfloat16*)b, (__hip_bfloat16*)c);
}
